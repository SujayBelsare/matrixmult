#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <memory>
#include <cstdint>
#include <filesystem>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

// Optimized block size - increased for better occupancy
#define BLOCK_DIM 256
// Larger batch size for better amortization of launch overhead
#define BATCH_SIZE 64

// Improved kernel using warp-level operations and better memory access patterns
__global__ void computeHistogramKernel(const int *__restrict__ input,
                                       unsigned int *__restrict__ histogram,
                                       int N, int B, int offset, int elements_to_process)
{
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();

    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    const unsigned int lane_id = threadIdx.x % 32;
    const unsigned int warp_id = threadIdx.x / 32;

    // Shared memory for local histogram
    extern __shared__ unsigned int s_hist[];

// Initialize shared memory histogram with more efficient loop
#pragma unroll 4
    for (int i = threadIdx.x; i < B; i += blockDim.x)
    {
        s_hist[i] = 0;
    }

    block.sync();

    // Process input data with coalesced memory access pattern
    for (int i = tid; i < elements_to_process; i += stride)
    {
        int value = input[i + offset];

        // Ensure value is within bounds of the histogram
        if (value >= 0 && value < B)
        {
            atomicAdd(&s_hist[value], 1u);
        }
    }

    block.sync();

    // Merge local histogram into global histogram with warp-level aggregation
    // to reduce atomic contention
    for (int i = threadIdx.x; i < B; i += blockDim.x)
    {
        unsigned int val = s_hist[i];
        if (val > 0)
        {
            atomicAdd(&histogram[i], val);
        }
    }
}

namespace solution
{
    std::string compute(const std::string &input_path, int N, int B)
    {
        std::string sol_path = std::filesystem::temp_directory_path() / "student_histogram.dat";

        // Select GPU device and get device properties
        hipSetDevice(0);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        // Calculate optimal parameters based on device properties
        const int elements_per_batch = BATCH_SIZE * BLOCK_DIM * 1024;
        const int num_batches = (N + elements_per_batch - 1) / elements_per_batch;

        // Use 3 streams for better overlapping of operations
        hipStream_t streams[3];
        for (int i = 0; i < 3; i++)
        {
            hipStreamCreate(&streams[i]);
        }

        // Configure kernel execution parameters - dynamically adjust based on device
        const int threadsPerBlock = BLOCK_DIM;
        // Use more blocks to increase parallelism, but limit to max supported
        const int blocks = std::min(deviceProp.multiProcessorCount * 32,
                                    (elements_per_batch + threadsPerBlock - 1) / threadsPerBlock);

        // Allocate device memory for histogram and zero it
        unsigned int *d_histogram;
        hipMalloc(&d_histogram, sizeof(unsigned int) * B);
        hipMemsetAsync(d_histogram, 0, sizeof(unsigned int) * B, streams[0]);

        // Use pinned memory for host buffers to enable faster transfer
        int *h_input_buffers[2];
        hipHostMalloc(&h_input_buffers[0], sizeof(int) * elements_per_batch);
        hipHostMalloc(&h_input_buffers[1], sizeof(int) * elements_per_batch);

        // Allocate device memory for input - double buffering
        int *d_input_buffers[2];
        hipMalloc(&d_input_buffers[0], sizeof(int) * elements_per_batch);
        hipMalloc(&d_input_buffers[1], sizeof(int) * elements_per_batch);

        // Final histogram will be stored here
        unsigned int *h_histogram;
        hipHostMalloc(&h_histogram, sizeof(unsigned int) * B);

        // Open input file
        std::ifstream input_fs(input_path, std::ios::binary);
        if (!input_fs)
        {
            std::cerr << "Error: Could not open input file " << input_path << std::endl;
            // Cleanup resources before returning
            hipHostFree(h_input_buffers[0]);
            hipHostFree(h_input_buffers[1]);
            hipHostFree(h_histogram);
            hipFree(d_input_buffers[0]);
            hipFree(d_input_buffers[1]);
            hipFree(d_histogram);
            for (int i = 0; i < 3; i++)
            {
                hipStreamDestroy(streams[i]);
            }
            return "";
        }

        // Process data in batches with triple buffering strategy
        // Start with first batch read & transfer
        hipEvent_t events[num_batches];
        for (int i = 0; i < num_batches; i++)
        {
            hipEventCreate(&events[i]);
        }

        int prev_buffer_idx = -1;
        for (int batch = 0; batch < num_batches; batch++)
        {
            int buffer_idx = batch % 2;
            int offset = batch * elements_per_batch;
            int elements_this_batch = std::min(elements_per_batch, N - offset);
            size_t bytes_to_read = elements_this_batch * sizeof(int);

            // Read batch of input data
            input_fs.read(reinterpret_cast<char *>(h_input_buffers[buffer_idx]), bytes_to_read);

            // Use events for better synchronization
            int stream_idx = batch % 3;

            // Transfer batch to device
            hipMemcpyAsync(d_input_buffers[buffer_idx], h_input_buffers[buffer_idx],
                            bytes_to_read, hipMemcpyHostToDevice, streams[stream_idx]);

            // Process previous batch while current batch is being transferred
            if (prev_buffer_idx >= 0)
            {
                // Previous batch should be in the device by now
                int prev_offset = (batch - 1) * elements_per_batch;
                int prev_elements = std::min(elements_per_batch, N - prev_offset);
                int prev_stream_idx = (batch - 1) % 3;

                // Use correct shared memory size based on data type
                size_t shared_mem_size = sizeof(unsigned int) * B;

                // Launch kernel with actual offset parameter
                computeHistogramKernel<<<blocks, threadsPerBlock, shared_mem_size, streams[prev_stream_idx]>>>(
                    d_input_buffers[prev_buffer_idx], d_histogram, N, B, 0, prev_elements);

                // Record event for this batch completion
                hipEventRecord(events[batch - 1], streams[prev_stream_idx]);
            }

            prev_buffer_idx = buffer_idx;
        }

        // Process the last batch
        if (num_batches > 0)
        {
            int last_buffer_idx = (num_batches - 1) % 2;
            int last_offset = (num_batches - 1) * elements_per_batch;
            int last_elements = std::min(elements_per_batch, N - last_offset);
            int last_stream_idx = (num_batches - 1) % 3;

            size_t shared_mem_size = sizeof(unsigned int) * B;
            computeHistogramKernel<<<blocks, threadsPerBlock, shared_mem_size, streams[last_stream_idx]>>>(
                d_input_buffers[last_buffer_idx], d_histogram, N, B, 0, last_elements);

            hipEventRecord(events[num_batches - 1], streams[last_stream_idx]);
        }

        // Wait for all computations to complete by waiting for last event
        if (num_batches > 0)
        {
            hipEventSynchronize(events[num_batches - 1]);
        }

        // Copy final histogram from device to host using stream 0
        hipMemcpyAsync(h_histogram, d_histogram, sizeof(unsigned int) * B,
                        hipMemcpyDeviceToHost, streams[0]);
        hipStreamSynchronize(streams[0]);

        // Write output - convert to int for file format compatibility
        std::ofstream sol_fs(sol_path, std::ios::binary);
        if (!sol_fs)
        {
            std::cerr << "Error: Could not open output file " << sol_path << std::endl;
            // Cleanup resources before returning
            hipHostFree(h_input_buffers[0]);
            hipHostFree(h_input_buffers[1]);
            hipHostFree(h_histogram);
            hipFree(d_input_buffers[0]);
            hipFree(d_input_buffers[1]);
            hipFree(d_histogram);
            for (int i = 0; i < 3; i++)
            {
                hipStreamDestroy(streams[i]);
            }
            for (int i = 0; i < num_batches; i++)
            {
                hipEventDestroy(events[i]);
            }
            return "";
        }

        auto int_histogram = std::make_unique<int[]>(B);
        for (int i = 0; i < B; i++)
        {
            int_histogram[i] = static_cast<int>(h_histogram[i]);
        }
        sol_fs.write(reinterpret_cast<const char *>(int_histogram.get()), sizeof(int) * B);
        sol_fs.close();

        // Proper cleanup of all resources
        hipHostFree(h_input_buffers[0]);
        hipHostFree(h_input_buffers[1]);
        hipHostFree(h_histogram);
        hipFree(d_input_buffers[0]);
        hipFree(d_input_buffers[1]);
        hipFree(d_histogram);
        for (int i = 0; i < 3; i++)
        {
            hipStreamDestroy(streams[i]);
        }
        for (int i = 0; i < num_batches; i++)
        {
            hipEventDestroy(events[i]);
        }

        return sol_path;
    }
}